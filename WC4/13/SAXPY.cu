
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
  int N = 1 << 20; // 1 million elements
  float *x, *y, *d_x, *d_y;

  // Allocate host memory
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));

  // Initialize host arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Allocate device memory
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  // Copy data from host to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch SAXPY kernel on the GPU
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  // Copy result back to host
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  // Verify result
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i] - 4.0f));
  }
  printf("Max error: %f\n", maxError);

  // Free memory
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}
