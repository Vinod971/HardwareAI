#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    for (int power = 15; power <= 25; ++power) {
        int N = 1 << power;
        size_t size = N * sizeof(float);
        float *x, *y, *d_x, *d_y;

        // Allocate host memory
        x = (float*)malloc(size);
        y = (float*)malloc(size);

        // Initialize host arrays
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        // Allocate device memory
        hipMalloc(&d_x, size);
        hipMalloc(&d_y, size);

        // Copy data from host to device
        hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

        // Setup CUDA events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Launch SAXPY kernel and time it
        hipEventRecord(start);
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
        hipEventRecord(stop);

        // Wait for kernel to finish and measure elapsed time
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Output result
        printf("N = 2^%d (%d): Execution Time = %.3f ms\n", power, N, milliseconds);

        // Cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
    }

    return 0;
}
