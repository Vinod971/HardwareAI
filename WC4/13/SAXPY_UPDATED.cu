#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

void run_saxpy(int N) {
    float *x, *y, *d_x, *d_y;
    hipEvent_t start, stop, memoryStart, memoryStop, kernelStart, kernelStop;

    // Allocate host memory
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Create events to measure time
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&memoryStart);
    hipEventCreate(&memoryStop);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    // Record the start event
    hipEventRecord(start);

    // Measure memory allocation and transfer time
    hipEventRecord(memoryStart);

    // Copy data from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(memoryStop);
    hipEventSynchronize(memoryStop);

    // Measure kernel execution time
    hipEventRecord(kernelStart);

    // Launch SAXPY kernel on the GPU
    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed times
    float totalMilliseconds = 0;
    float memoryMilliseconds = 0;
    float kernelMilliseconds = 0;

    hipEventElapsedTime(&totalMilliseconds, start, stop);
    hipEventElapsedTime(&memoryMilliseconds, memoryStart, memoryStop);
    hipEventElapsedTime(&kernelMilliseconds, kernelStart, kernelStop);

    printf("Execution time for N=%d:\n", N);
    printf("  Total time: %f ms\n", totalMilliseconds);
    printf("  Memory transfer time: %f ms\n", memoryMilliseconds);
    printf("  GPU computation time (kernel): %f ms\n", kernelMilliseconds);

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(memoryStart);
    hipEventDestroy(memoryStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
}

int main(void)
{
    // Matrix sizes: N = 2^15, 2^16, ..., 2^25
    int sizes[] = { 1 << 15, 1 << 16, 1 << 17, 1 << 18, 1 << 19, 1 << 20, 1 << 21, 1 << 22, 1 << 23, 1 << 24, 1 << 25 };
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    // Run SAXPY for each matrix size
    for (int i = 0; i < num_sizes; i++) {
        run_saxpy(sizes[i]);
    }

    return 0;
}
