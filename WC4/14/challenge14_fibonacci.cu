
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// CPU Fibonacci
void fibonacci_cpu(int* fib, int n) {
    fib[0] = 0;
    fib[1] = 1;
    for (int i = 2; i < n; ++i) {
        fib[i] = fib[i - 1] + fib[i - 2];
    }
}

// GPU Fibonacci Kernel
__global__ void fibonacci_gpu(int* fib, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (i == 0) fib[i] = 0;
        else if (i == 1) fib[i] = 1;
        else {
            int a = 0, b = 1, c;
            for (int j = 2; j <= i; ++j) {
                c = a + b;
                a = b;
                b = c;
            }
            fib[i] = b;
        }
    }
}

int main() {
    const int N = 1000;
    int* h_fib_cpu = new int[N];
    int* h_fib_gpu = new int[N];

    // CPU Benchmark
    auto start_cpu = std::chrono::high_resolution_clock::now();
    fibonacci_cpu(h_fib_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double time_cpu = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();

    // Allocate device memory
    int* d_fib;
    hipMalloc((void**)&d_fib, N * sizeof(int));

    // GPU Benchmark
    auto start_gpu = std::chrono::high_resolution_clock::now();
    fibonacci_gpu<<<(N + 255)/256, 256>>>(d_fib, N);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    double time_gpu = std::chrono::duration<double, std::milli>(end_gpu - start_gpu).count();

    // Copy results back to host
    hipMemcpy(h_fib_gpu, d_fib, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify correctness
    bool match = true;
    for (int i = 0; i < N; ++i) {
        if (h_fib_cpu[i] != h_fib_gpu[i]) {
            match = false;
            break;
        }
    }

    std::cout << "CPU Time: " << time_cpu << " ms\n";
    std::cout << "GPU Time: " << time_gpu << " ms\n";
    std::cout << "Results Match: " << (match ? "Yes" : "No") << std::endl;

    // Cleanup
    delete[] h_fib_cpu;
    delete[] h_fib_gpu;
    hipFree(d_fib);

    return 0;
}
